#include "hip/hip_runtime.h"
// matrixMul_kernel.cu
//
//	Matrix multiplication: C = A * B.
// 
//

#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "CH\chTimer.h"
#include <hipblas.h>

#define block_size 32

/*
#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) cutilBankChecker(((float*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) cutilBankChecker(((float*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif
*/

////////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: C = A * B (No shared memory used)
//! hA is A's height, wA is A's width and wB is B's width 
//! In other words, it computes hAxwA matrix multiplied by wAxWB matrix.
//! Assume matrices are stored in row-major linear array, and matrix indexing is 0-based.
////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void
matrixMul_v1( float* C, float* A, float* B, int hA, int wA, int wB)
{
    // index of the C matrix element computed by this thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
    // Cvalue is used to store the element of the C matrix computed by this thread
	float Cvalue;
    if (row < hA && col < wB)
    {
		Cvalue = 0.0f;

		// Loop over all the matrices of A and B required to compute C
		for (int k = 0; k < wA; k++) 
			Cvalue += A[row * wA + k] * B[k * wB + col];

		// Write the block sub-matrix to device memory; each thread writes one element
		C[row * wB + col] = Cvalue;
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: C = A * B (shared memory used)
//! hA is A's height, wA is A's width and wB is B's width 
//! In other words, it computes hAxwA matrix multiplied by wAxWB matrix.
//! Assume matrices are stored in row-major linear array, and matrix indexing is 0-based.
////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void
matrixMul_v2( float* C, float* A, float* B, int hA, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * block_size  * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = block_size;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = block_size * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = block_size * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) 
    {
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[block_size][block_size];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[block_size][block_size];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < block_size; ++k)
            Csub += As[ty][k] * Bs[k][tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * block_size * by + block_size * bx;
    C[c + wB * ty + tx] = Csub;
}


/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
/*
template <int block_size> __global__ void
matrixMulCUDA(float *C, float *A, float *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * block_size * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = block_size;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = block_size * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = block_size * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[block_size][block_size];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[block_size][block_size];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < block_size; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * block_size * by + block_size * bx;
    C[c + wB * ty + tx] = Csub;
}


int matrixMultiply(int argc, char **argv, int block_size, dim3 &dimsA, dim3 &dimsB)
{
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // Initialize host memory
    const float valB = 0.01f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    hipError_t error;

    error = hipMalloc((void **) &d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16)
    {
        matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
    else
    {
        matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

    printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++)
    {
        if (block_size == 16)
        {
            matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
    }

    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula 
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps 
    double eps = 1.e-6 ; // machine zero
    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err/abs_val/dot_length ;
        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("\nNote: For peak performance, please refer to the matrixMulCUBLAS example.\n");

    hipDeviceReset();

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}

*/

//
// Our CUDA matrix multiplication interface function
//
int GPU_SGEMM (float* C, float* A, float* B, int HA, int WA, int WB, int version)		
{
    float *d_A = 0;
    float *d_B = 0;
    float *d_C = 0;
    // float alpha = 1.0f;
    // float beta = 0.0f;

	hipError_t cudaStatus;

	// Make sure CUDA device 0 is available
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }
 
    /* Allocate device memory for the matrices (d_A, d_B, and d_C) */
    if (hipMalloc((void **)&d_A, HA * WA * sizeof(d_A[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }
    if (hipMalloc((void **)&d_B, WA * WB * sizeof(d_B[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
        return EXIT_FAILURE;
    }
    if (hipMalloc((void **)&d_C, HA * WB * sizeof(d_C[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
        return EXIT_FAILURE;
    }
	// Copy host memory (A, and B) to device
    cudaStatus = hipMemcpy(d_A, A, HA*WA*sizeof(A[0]), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("hipMemcpy (d_A, A) returned error code %d\n", cudaStatus);
        exit(EXIT_FAILURE);
    }
    cudaStatus = hipMemcpy(d_B, B, WA*WB*sizeof(B[0]), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("hipMemcpy (d_B, B) returned error code %d\n", cudaStatus);
        exit(EXIT_FAILURE);
    }

	// Setup execution parameters and call kernel
    dim3 block(block_size, block_size);
    dim3 grid ((WB+block_size-1)/block_size, (HA+ block_size-1)/block_size);
	if (version == 1)
		matrixMul_v1<<< grid, block >>>(d_C, d_A, d_B, HA, WA, WB); 
	else 
		matrixMul_v2<<< grid, block >>>(d_C, d_A, d_B, HA, WA, WB);   
    hipDeviceSynchronize();

	// Copy result (C)  from device to host
    cudaStatus = hipMemcpy(C, d_C, HA*WB*sizeof(C[0]), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        printf("hipMemcpy (C, d_C) returned error code\n", cudaStatus);
        exit(EXIT_FAILURE);
    }

    /* Device memory clean up */
    if (hipFree(d_A) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_B) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (B)\n");
        return EXIT_FAILURE;
    }
    if (hipFree(d_C) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (C)\n");
        return EXIT_FAILURE;
    }

	return 0;
}
